#include <iostream>
#include <glad/glad.h>

#include <hip/hip_runtime.h>
#include <cuda_gl_interop.h>

#include "particle_renderer.hpp"
#include "nbody.cuh"

#define DIM 512

hipGraphicsResource *resource;

void setDevice() {
    hipDeviceProp_t prop;
    int dev;

    memset(&prop, 0, sizeof(hipDeviceProp_t));
    prop.major = 1;
    prop.minor = 0;
    hipChooseDevice(&dev, &prop);

    cudaGLSetGLDevice(dev);
}
